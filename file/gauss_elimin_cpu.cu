

#include <hip/hip_runtime.h>
#include <bits/stdc++.h>
#include <time.h>
#include <sys/time.h>
using namespace std;
#define N 20

void make_primary_matrix(double arr[], int n) {
	for (int i=0; i<n; i++) {
		for (int j=i; j<n; j++) {
			arr[i*(n+1)+j] = arr[j*(n+1)+i] = i+1;
		}
	}
	for (int i=0; i<n; i++) {
		int num = 0;
		for (int j=0; j<n; j++) {
			num += arr[i*(n+1)+j];
		}
		arr[i*(n+1)+n] = num;
	}
}

void swap_arr(double arr[], int col1, int col2, int n) {
	for (int i=0; i<n+1; i++) {
		swap(arr[(n+1)*col1 + i], arr[(n+1)*col2 + i]);
	}
}

void gaussElimin(double arr[], double *x, int n) {
	for (int j=0; j<n-1; j++) {
		if (abs(arr[j*(n+1)+j] - 0.0) < 1e-10) {
			for (int k=j+1; k<n; k++) {
				if (abs(arr[k*(n+1)+j] - 0.0) > 1e-10) {
					swap_arr(arr, k, j, n);
					break;
				}
			}
		}

		for (int i=j+1; i<n; i++) {
			double lam = arr[i*(n+1)+j] / arr[j*(n+1)+j];
			for (int k=0; k<n; k++) {
				arr[i*(n+1)+k] -= lam*arr[j*(n+1)+k];
			}
			arr[i*(n+1)+n] -= lam*arr[j*(n+1)+n];
		}
	}

	// 역대입법
	x[n-1] = arr[(n-1)*(n+1)+n] / arr[(n-1)*(n+1)+n-1];

	for (int i=n-2; i>=0; i--) {
		double val = 0;
		for (int k=i+1; k<n; k++) {
			val += x[k] * arr[i*(n+1)+k];
		}
		x[i] = (arr[i*(n+1)+n] - val) / arr[i*(n+1)+i];
	}

}

int main()
{	
    struct timeval start, end;


	double *arr, *x;
	arr = (double*) malloc((N)*(N+1)*sizeof(double));
    x = (double*) malloc(N*sizeof(double));

    printf("size of matrix:  %d\n", N);

    make_primary_matrix(arr, N);

    gettimeofday(&start, NULL);
    gaussElimin(arr, x, N);
    gettimeofday(&end, NULL);

    double start_micro = (double)start.tv_sec*1000000 + (double)start.tv_usec;
    double end_micro = (double)end.tv_sec*1000000 + (double)end.tv_usec;

    printf("\n%lf\n", (end_micro-start_micro)/1000000);

	return 0;
}
