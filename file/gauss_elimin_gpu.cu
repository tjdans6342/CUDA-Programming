

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <sys/time.h>
#define N 20

#define THREADS_PER_BLOCK 16

// Host function -------------------------------------------------------------------
double *malloc_matrix(const int a, const int b) {
    return (double*)malloc(sizeof(double *)*a*b);
}

double *make_primary_matrix(int *a_size) { // 해가 유일한 행렬 만들기
    
    int size = N;
    double *matrix_arr = malloc_matrix(size, size + 1);

    for (int i=0; i<N; i++) {
        for (int j=i; j<N; j++) {
            matrix_arr[i*(N+1)+j] = matrix_arr[j*(N+1)+i] = i+1;
        }
    }
    for (int i=0; i<N; i++) {
        int num = 0;
        for (int j=0; j<N; j++) {
            num += (j+1)*matrix_arr[i*(N+1)+j];
        }
        matrix_arr[i*(N+1)+N] = num;
    }

    *a_size = size;

    return matrix_arr;
}
// --------------------------------------------------------------------------------------


// Device function -------------------------------------------------------------------
__global__ void replace_zero_gpu(double *d_arr, int rows, int columns, int column) {
    if(fabs(d_arr[column*columns + column]) <= 1e-4) {
        int row = column;
        for(; row < rows; row++) {
            if(fabs(d_arr[row*columns + column]) > 1e-4)
                break;
        }
        int tidx= blockDim.x*blockIdx.x + threadIdx.x;
        if(tidx+ column >= columns)
            return;

        __syncthreads();
        int zero = column*columns + column + tidx; // x축 이동
        int chosen = row*columns + column + tidx; // x축 이동
        d_arr[zero] += d_arr[chosen];
    }
}

__global__ void column_elimination_gpu(double *d_arr, int rows, int columns, int col) {
    int tidx= blockDim.x*blockIdx.x + threadIdx.x;
    if(tidx >= (rows - 1 - col)*(columns - col)) // columns 만큼의 thread를 사용하지 x
        return;

    int sub_y = tidx/(columns-col);
    int sub_x = tidx%(columns-col);
    int gl_y = col+1 + sub_y;
    int gl_x = col + sub_x;

    int gl_idx = gl_x + gl_y*columns;
    int up_idx = gl_x + col*columns;

    int up_el = col + col*columns;
    int gl_el = col + gl_y*columns;
    double lam = d_arr[gl_el]/d_arr[up_el];

    d_arr[gl_idx] -= lam*d_arr[up_idx];
}

__global__ void multiple_column(double *d_arr, int rows, int columns, int row) {
    int tidx= threadIdx.x;

    int cols = columns - 2 - row; // 바꿔야 하는 개수

    int start_index_cols = row*columns + row;
    int end_index_rows = rows*columns - 1;

    d_arr[start_index_cols + tidx+1] *= d_arr[end_index_rows - columns*(cols-1-tidx)];
}

__global__ void reverse_row_elimination(double *d_arr, int rows, int columns, int row) {
    int tidx= threadIdx.x;
    int cols = columns - 2 - row;

    int start_index = row*columns + row; // (row, row) 좌표

    for (int i=cols; i>=2; i/=2) {
        bool is_odd;
        if (i%2 == 1) is_odd = true;
        else is_odd = false;

        int step = i/2;
        if (tidx>= step) return;

        d_arr[start_index + tidx+1] += (d_arr[start_index + tidx+1 + step]); 
        d_arr[start_index + tidx+1 + step] = 0;

        if (is_odd && tidx+1 == step) {
            d_arr[start_index + tidx+1] += d_arr[start_index + tidx+1 + step+1];
            d_arr[start_index + tidx+1 + step+1] = 0;
        }

        __syncthreads();
    }

    int x_el = (row + 1)*columns - 1;
    int diag_el = row*columns + row;

    if(diag_el + 1 != x_el) {
        d_arr[x_el] -= d_arr[diag_el + 1];
        d_arr[diag_el + 1] = 0.0;
    }

    d_arr[x_el] /= d_arr[diag_el];
    d_arr[diag_el] = 1.0;
}

// -----------------------------------------------------------------------------------


// Host function -------------------------------------------------------------------
void start_gaussian_elimination_gpu(double *arr, int rows, int cols) {
    double *dev_arr;

    hipMalloc(&dev_arr, sizeof(double)*rows*cols);
    hipMemcpy(dev_arr, (void*)arr, sizeof(double)*rows*cols, hipMemcpyHostToDevice);

    int block_size;

    for(int y=0; y<cols-1; y++) {
        block_size = (cols-y-1)/THREADS_PER_BLOCK + 1;
        replace_zero_gpu <<<block_size, THREADS_PER_BLOCK>>> (dev_arr, rows, cols, y);

        block_size = ((rows-1 - y )*(cols - y) - 1)/THREADS_PER_BLOCK + 1;
        column_elimination_gpu <<<block_size, THREADS_PER_BLOCK>>> (dev_arr, rows, cols, y);
    }

    for(int x=rows-1; x>=0; x--) {

        multiple_column<<<1, cols-2-x>>>(dev_arr, rows, cols, x);
        reverse_row_elimination<<<1, cols>>>(dev_arr, rows, cols, x);

    }

    hipMemcpy(arr, (void*)dev_arr, sizeof(double)*rows*cols, hipMemcpyDeviceToHost);
    for (int i=0; i<rows; i++) {
        for (int j=0; j<cols; j++) {
            printf("%.2f ", arr[i*cols+j]);
        }
        printf("\n");
    }
    printf("\n");
    
    hipFree(dev_arr);
}


int main(int argc, char ** argv) {
    struct timeval start, end;

    int size;
    double *arr = make_primary_matrix(&size);

    printf("size of matrix:  %d\n", N);

    gettimeofday(&start, NULL);
    start_gaussian_elimination_gpu(arr, size, size + 1);
    gettimeofday(&end, NULL);


    double start_micro = (double)start.tv_sec*1000000 + (double)start.tv_usec;
    double end_micro = (double)end.tv_sec*1000000 + (double)end.tv_usec;

    printf("\n%lf\n", (end_micro-start_micro)/1000000);

    return 0;
}
// ---------------------------------------------------------------------------------------
